#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"

#define TAM_BLOCO 256
#define FATOR 1e-9f



void setaPosicao(float *data, int corpos) {
	for (int i = 0; i < 4 * corpos; i++) {
		data[i] = 2.0f * (rand() / (float)RAND_MAX) - 1.0f;
	}
}
void zeraAceleracao(float *data, int corpos) {
	for (int i = 4 * corpos; i < 8 * corpos; i++) {
		data[i] = 0.0f;
	}
}


__device__ float3 tile_aceleracao(float4 minhaPosicao, float3 acel){
	int i;
	//Cria link com os dados da memoria compartilhada
	extern __shared__ float4 posCompartilhada[];
	//Faz o calculo em rela��o a todos os corpos do bloco
	for (i = 0; i < TAM_BLOCO; i++) {
		float3 r;
		//calcula a distancia entre os corpos
		r.x = posCompartilhada[i].x - minhaPosicao.x;
		r.y = posCompartilhada[i].y - minhaPosicao.y;
		r.z = posCompartilhada[i].z - minhaPosicao.z;
		//modulo da distancia ao quadrado + fator de amaciamento
		float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + FATOR;
		//eleva a 6 potencia
		float dist_a_6 = distSqr * distSqr * distSqr;
		//tira a raiz quadrada e inverte o resultado
		float invDistanciaCubo = 1.0f / sqrtf(dist_a_6);
		//multiplica pela massa do corpo
		float s = posCompartilhada[i].w * invDistanciaCubo;
		//multiplica pelo vetor distancia e realiza mais 1 soma do 
		//somatorio
		acel.x += r.x * s;
		acel.y += r.y * s;
		acel.z += r.z * s;
	}
	return acel;
}

__global__ void calcula_aceleracao(void *devX, void *devA, int ncorpos, int nBlocos){
	//Cria link com os dados da memoria compartilhada
	extern __shared__ float4 posCompartilhada[];
	//Cria vetor com as posicoes na memoria global
	float4 *globalX = (float4 *)devX;
	//Cria vetor com as aceleracoes na memoria global
	float4 *globalA = (float4 *)devA;
	float4 minhaPosicao;
	int i, tile;
	float3 acc = { 0.0f, 0.0f, 0.0f };
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	minhaPosicao = globalX[gtid];
	//Faz o calculo em rela��o a todos os blocos da grid
	for (tile = 0; tile < gridDim.x; tile++) {
		int idx = tile * blockDim.x + threadIdx.x;
		//Carrega as posicoes da memoria global para a compartilhada
		posCompartilhada[threadIdx.x] = globalX[idx];
		__syncthreads();
		acc = tile_aceleracao(minhaPosicao, acc);
		__syncthreads();
	}
	// salva o resultado na memoria global para o passo da integracao.  
	float4 acc4 = { acc.x, acc.y, acc.z, 0.0f };
	globalA[gtid] = acc4;
}


int main(const int argc, const char** argv) {
	int i;//i=aplicacao corrente
	float score = 0;
	int nquad;
	int nBodies, nIters;
	float temp0, temp1, temp2, temp3, temp4, temp5;
	int qntCores, frequenciaGpu;
	double eficiencia;

	for (i = 0; i < 6; i++){

		if (i == 0){
			nBodies = 1024;
			nIters = 1000;
		}
		else if (i == 1){
			nBodies = 2048;
			nIters = 1000;
		}
		else if (i == 2){
			nBodies = 4096;
			nIters = 1000;
		}
		else if (i == 3){
			nBodies = 8192;
			nIters = 1000;
		}
		else if (i == 4){
			nBodies = 16384;
			nIters = 500;
		}
		else if (i == 5){
			nBodies = 32768;
			nIters = 250;
		}
		nquad = nBodies*nBodies;

		const float dt = 0.01f; // tempo

		int bytes = 2 * nBodies*sizeof(float4);
		float *buf = (float*)malloc(bytes);
		float4 *bp = (float4*)buf;
		float4 *ba = ((float4*)buf) + nBodies;

		setaPosicao(buf, nBodies); // Inicia posicao e acel
		zeraAceleracao(buf, nBodies);

		float *d_buf;
		hipMalloc(&d_buf, bytes);
		float4 *d_bp = (float4*)d_buf;
		float4 *d_ba = ((float4*)d_buf) + nBodies;

		int nBlocks = (nBodies + TAM_BLOCO - 1) / TAM_BLOCO;
		double totalTime = 0.0;

		for (int iter = 1; iter <= nIters; iter++) {
			StartTimer();

			//Passa os dados da memoria para a GPU
			hipMemcpy(d_buf, buf, bytes, hipMemcpyHostToDevice);
			//Calcula a aceleracao de cada corpo (cada thread cuida de um corpo)
			//O terceiro parametro na chamada da funcao em cuda e o tamanho da memoria compartilhada
			calcula_aceleracao << <nBlocks, TAM_BLOCO, (TAM_BLOCO*sizeof(float4)) >> >(d_bp, d_ba, nBodies, nBlocks);
			//Passa os dados da GPU para a memoria
			hipMemcpy(buf, d_buf, bytes, hipMemcpyDeviceToHost);

			//Faz a integracao das posicoes
			for (int i = 0; i < nBodies; i++) {
				bp[i].x += ba[i].x*dt*dt;
				bp[i].y += ba[i].y*dt*dt;
				bp[i].z += ba[i].z*dt*dt;


			}

			const double tElapsed = GetTimer() / 1000.0;
			if (iter > 1) {
				totalTime += tElapsed;
			}

			printf("Ciclo %d Iteracao %d: %.3f segundos\n", i, iter, tElapsed);

		}
		double avgTime = totalTime / (double)(nIters - 1);
		score += (nquad / totalTime);



		free(buf);
		hipFree(d_buf);

		if (i == 0) temp0 = totalTime;
		if (i == 1) temp1 = totalTime;
		if (i == 2) temp2 = totalTime;
		if (i == 3) temp3 = totalTime;
		if (i == 4) temp4 = totalTime;
		if (i == 5) temp5 = totalTime;
	}
	//qntCores = 384;
	//frequenciaGpu = 980;
	printf("\nInsira a quantidade de cuda cores presentes em sua GPU\n");
	scanf("%d", &qntCores);
	printf("\nInsira a frequencia de sua GPU em MHz\n");
	scanf("%d", &frequenciaGpu);
	eficiencia = (1.0 / qntCores)*(500.0 / frequenciaGpu)*score;

	printf("\n Score %f\n", score);
	printf("\n Eficiencia %f\n", eficiencia);

	printf("\nTempo ciclo 0 %f", temp0);
	printf("\nTempo ciclo 1 %f", temp1);
	printf("\nTempo ciclo 2 %f", temp2);
	printf("\nTempo ciclo 3 %f", temp3);
	printf("\nTempo ciclo 4 %f", temp4);
	printf("\nTempo ciclo 5 %f\n", temp5);
}
